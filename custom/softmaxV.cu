
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const size_t DSIZE = 16384;      // matrix side dimension
// const size_t DSIZE = 1024;      // matrix side dimension
const int block_size = 1024;  // CUDA maximum is 1024
const float element_val = 5;
const float Dim = 512;
const float v_val = 1;


__global__ void softmax(float *A, float *sums, size_t ds){

  int idx = threadIdx.x;
  __shared__ float sdata[block_size];
  sdata[idx] = 0.0f;
  float val = 0;

  for(int i = 0; i < ds/blockDim.x; i++){
    val = expf(A[ds*blockIdx.x + i*blockDim.x + idx]);
    A[ds*blockIdx.x + i*blockDim.x + idx] = val;
    sdata[idx] += val;
  }
  
  for(int s = blockDim.x/2; s > 0; s/=2){
    __syncthreads();
    if (idx < s) sdata[idx] += sdata[idx + s];
  }
  
  if (idx == 0) sums[blockIdx.x] = sdata[0];
  
  for(int i = 0; i < ds/blockDim.x; i++) A[ds*blockIdx.x + i*blockDim.x + idx] /= sdata[0];
  
}

__global__ void softmaxV(float *QK, float *V, float *ACT, size_t ds){

  int idx = threadIdx.x;
  __shared__ float sdata[block_size];
  float val;
  sdata[idx] = 0.0f;

  for(int i = 0; i < ds/blockDim.x; i++){
    val = expf(QK[ds*blockIdx.x + i*blockDim.x + idx]);
    QK[ds*blockIdx.x + i*blockDim.x + idx] = val;
    sdata[idx] += val;
  }
  
  for(int s = blockDim.x/2; s > 0; s/=2){
    __syncthreads();
    if (idx < s) sdata[idx] += sdata[idx + s];
  }
  
  for(int i = 0; i < ds/blockDim.x; i++) QK[ds*blockIdx.x + i*blockDim.x + idx] /= sdata[0];


  
}

bool validate(float *data, size_t sz){
  
  for (size_t i = 0; i < sz; i++){
    // printf("%f\n", expf(0.005)*(float)sz);
    float val = expf(element_val)*(float)sz;
    if (data[i] - val > 0.001) {printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], val); return false;}
  }
    return true;
}
int main(){

    float *h_QK, *h_V;
    float *d_QK, *d_V, *d_ACT;

    h_QK = new float[DSIZE*DSIZE];  // allocate space for data in host memory
    // h_V = new float[DSIZE*Dim];

    for (int i = 0; i < DSIZE*DSIZE; i++)  h_QK[i] = element_val;

    for (int i = 0; i < DSIZE*Dim; i++) h_V[i] = v_val;

    hipMalloc(&d_QK, DSIZE*DSIZE*sizeof(float));  // allocate device space for A
    // cudaMalloc(&d_V, DSIZE*Dim*sizeof(float));  // allocate device space for vector d_sums
    hipMalloc(&d_ACT, DSIZE*Dim*sizeof(float));  // allocate device space for vector d_sums

    cudaCheckErrors("hipMalloc failure"); // error checking
    hipMemcpy(d_QK, h_QK, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    cudaCheckErrors("hipMalloc failure"); // error checking
    // cudaMemcpy(d_V, h_V, DSIZE*Dim*sizeof(float), cudaMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    softmax<<<DSIZE, block_size>>>(d_QK, d_ACT, DSIZE);
    cudaCheckErrors("kernel launch failure");

    // cudaMemcpy(h_sums, d_sums, DSIZE*sizeof(float), cudaMemcpyDeviceToHost);
    // cudaCheckErrors("1 kernel execution failure or cudaMemcpy H2D failure");

    // if (!validate(h_sums, DSIZE)) return -1; 
    // printf("row sums correct!\n");

    hipMemcpy(h_QK, d_QK, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");

    printf("%.15f\n", h_QK[0]);

    for(int i = 0; i < DSIZE*DSIZE; i++){
    // printf("%f\n %f\n", h_A[i], 1/(float)DSIZE);
    if(h_QK[i] - 1/(float)DSIZE > 0.00001
    ) {printf("results mismatch at %d, was: %.10f, should be: %.10f\n", i, h_QK[i], 1/float(DSIZE)); return -1;}
    }
    printf("softmax correct!\n");



    return 0;
}
  
