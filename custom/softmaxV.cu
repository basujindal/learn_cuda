#include "hip/hip_runtime.h"
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const int N = 6;      // matrix side dimension
const int Dim = 1024;
const int block_size = 32;  // CUDA maximum is 1024 *total* threads in block  
const int block_size_softmax = 1024;  // CUDA maximum is 1024 *total* threads in block


__global__ void softmax(float *QK, size_t ds){

  int idx = threadIdx.x;
  __shared__ float sdata[block_size_softmax];
  sdata[idx] = 0.0f;
  float val = 0;

  __shared__ float max_val;
  max_val = 0.0f;

  // Find the maximum value in the block

  for (int index = start_index + idx; index < end_index; index += blockDim.x) {
    if (index < ds*ds) sdata[idx] = max(A[index], sdata[idx]);
  }

  for(int s = blockDim.x/2; s > 0; s/=2){
    __syncthreads();
    if (idx < s) sdata[idx] = max(sdata[idx], sdata[idx + s]);
  }
  __syncthreads();

  if (idx == 0) max_val = sdata[0];
  __syncthreads();

  sdata[idx] = 0.0f;

  for(int i = 0; i < ds/blockDim.x; i++){
    val = expf(QK[ds*blockIdx.x + i*blockDim.x + idx]);
    QK[ds*blockIdx.x + i*blockDim.x + idx] = val;
    sdata[idx] += val;
  }
  
  for(int s = blockDim.x/2; s > 0; s/=2){
    __syncthreads();
    if (idx < s) sdata[idx] += sdata[idx + s];
  }
  __syncthreads();
  
  for(int i = 0; i < ds/blockDim.x; i++) QK[ds*blockIdx.x + i*blockDim.x + idx] /= sdata[0];

  if(idx == 0) printf("sdata[0]: %f\n", sdata[0]);
  
}

__global__ void softmax_max(float *A, size_t ds){

  int idx = threadIdx.x;
  __shared__ float sdata[block_size_softmax];
  sdata[idx] = 0.0f;
  float val = 0.0f;
  float max_val = 0.0f;

  for(int i = 0; i < ds/blockDim.x; i++){
    sdata[idx] = max(A[ds*blockIdx.x + i*blockDim.x + idx], sdata[idx]);
  }

  for(int s = blockDim.x/2; s > 0; s/=2){
    __syncthreads();
    if (idx < s) sdata[idx] = max(sdata[idx], sdata[idx + s]);
  }
  __syncthreads();

  if (idx == 0) max_val = sdata[0];
  sdata[idx] = 0.0f;

  __syncthreads();

  for(int i = 0; i < ds/blockDim.x; i++){
    val = expf(A[ds*blockIdx.x + i*blockDim.x + idx] - max_val);
    A[ds*blockIdx.x + i*blockDim.x + idx] = val;
    sdata[idx] += val;
  }

  for(int s = blockDim.x/2; s > 0; s/=2){
    __syncthreads();
    if (idx < s) sdata[idx] += sdata[idx + s];
  }
  __syncthreads();
  
  for(int i = 0; i < ds/blockDim.x; i++) A[ds*blockIdx.x + i*blockDim.x + idx] /= sdata[0];

  if (idx == 0) printf("A: %f\n", A[ds*blockIdx.x]);
  

}


__global__ void matmul(const float *Attn, const float *V, float *C, int Dim, int N) {

  // declare cache in shared memory
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];
  
  int col = threadIdx.x+blockDim.x*blockIdx.x; // create thread x index
  int row = threadIdx.y+blockDim.y*blockIdx.y; // create thread y index

  if ((row < N) && (col < Dim)){
    float temp = 0;

    for (int i = 0; i < N/block_size; i++) {

      // Load data into shared memory
      As[threadIdx.y][threadIdx.x] = Attn[row*N + (block_size*i + threadIdx.x)];
      Bs[threadIdx.y][threadIdx.x] = V[col + Dim*(block_size*i + threadIdx.y)];

      __syncthreads();

      for (int k = 0; k < block_size; k++) temp +=  As[threadIdx.y][k] * Bs[k][threadIdx.x]; // dot product of row and column

      __syncthreads();

    }
    // Write to global memory
    C[row*Dim+col] = temp;
  }
}


__global__ void QK_V(const float *QK, const float *V, float *C, int Dim, int N) {

  // declare cache in shared memory
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];
  
  int col = threadIdx.x+blockDim.x*blockIdx.x; // create thread x index
  int row = threadIdx.y+blockDim.y*blockIdx.y; // create thread y index

  if ((row < N) && (col < Dim)){
    float temp = 0, val, sum = 0;

    for (int i = 0; i < N/block_size; i++) {

      // Load data into shared memory
      As[threadIdx.y][threadIdx.x] = expf(QK[row*N + (block_size*i + threadIdx.x)]);
      Bs[threadIdx.y][threadIdx.x] = V[col + Dim*(block_size*i + threadIdx.y)];

      __syncthreads();

      for (int k = 0; k < block_size; k++){
        val = As[threadIdx.y][k];
      	temp +=  val * Bs[k][threadIdx.x]; // dot product of row and column
        sum+=val;
      }

      __syncthreads();

    }

    // Write to global memory
    C[row*Dim+col] = temp/sum;
  }
}


int validateQK_V(float *h_QK, float *h_V, float *h_ACT, int N, int Dim){

  float sums[N];

  for (int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      h_QK[i*N+j] = expf(h_QK[i*N+j]);
      sums[i] += h_QK[i*N+j];
    }
  }


  for(int i = 0; i < N; i++) for (int j = 0; j < N; j++) h_QK[i*N+j] /= sums[i];

  for (int i = 0; i < N; i++){
    for (int j = 0; j < Dim; j++){
      float temp = 0;
      for (int k = 0; k < N; k++) temp += h_QK[i*N+k]*h_V[k*Dim+j];

      if (temp - h_ACT[i*Dim+j] > 0.1) {
        printf("results mismatch at %d, was: %f, should be: %f\n", i*Dim+j, h_ACT[i*Dim+j], temp);
        return -1;
      }
    }
  }

  printf("softmax correct!\n");
  return 0;
}

int validateSoftmax(float *h_QK, float *h_sout, int N){

  float sums[N];
  for (int i = 0; i < N; i++) sums[i] = 0;

  for (int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      h_QK[i*N+j] = expf(h_QK[i*N+j]);
      sums[i] += h_QK[i*N+j];
    }
  }


  for(int i = 0; i < N; i++) for (int j = 0; j < N; j++) h_QK[i*N+j] /= sums[i];


  for (int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      if (h_QK[i*N+j] - h_sout[i*N+j] > 0.001) {
        printf("results mismatch at %d, was: %f, should be: %f\n", i*N+j, h_sout[i*N+j], h_QK[i*N+j]);
        return -1;
      }
    }
  }

  printf("softmax correct!\n");
  return 0;
}

int main(){

    float *h_QK, *h_V, *h_ACT, *h_sout;
    float *d_QK, *d_V, *d_ACT;

    h_QK = new float[N*N];
    h_V = new float[N*Dim];
    h_ACT = new float[N*Dim];
    h_sout = new float[N*N];

    for (int i = 0; i < N*N; i++) h_QK[i] = rand()/(float)RAND_MAX;
    for (int i = 0; i < N*Dim; i++) h_V[i] = rand()/(float)RAND_MAX;


    hipMalloc(&d_QK, N*N*sizeof(float));
    hipMalloc(&d_V, N*Dim*sizeof(float));  
    hipMalloc(&d_ACT, N*Dim*sizeof(float)); 


    cudaCheckErrors("hipMalloc failure"); // error checking
    hipMemcpy(d_QK, h_QK, N*N*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    cudaCheckErrors("hipMalloc failure"); // error checking
    hipMemcpy(d_V, h_V, N*Dim*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    
    dim3 block(block_size, block_size);
    dim3 grid((Dim+block.x-1)/block.x, (Dim+block.y-1)/block.y);

    // Fused QK_V
    // QK_V<<<grid, block>>>(d_QK, d_V, d_ACT, Dim, N);

    // // Softmax + QK_V
    
    softmax<<<N, block_size_softmax>>>(d_QK, Dim);
    cudaCheckErrors("kernel launch failure");

    hipMemcpy(h_sout, d_QK, N*N*sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");

    validateSoftmax(h_QK, h_sout, N);

    // matmul<<<grid, block>>>(d_QK, d_V, d_ACT, Dim, N);
    // cudaCheckErrors("kernel launch failure");

    // hipMemcpy(h_ACT, d_ACT, N*Dim*sizeof(float), hipMemcpyDeviceToHost);
    // cudaCheckErrors("hipMemcpy D2H failure");

    // // Validate softmax(QK)*V
    // validateQK_V(h_QK, h_V, h_ACT, N, Dim);

    return 0;
}
  
