
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const size_t DSIZE = 16384;      // matrix side dimension
// const size_t DSIZE = 1024;      // matrix side dimension
const int block_size = 1024;  // CUDA maximum is 1024
const float element_val = 5;

__global__ void row_sums(const float *A, float *sums, size_t ds){

  int idx = threadIdx.x;
  __shared__ float sdata[block_size];
  sdata[idx] = 0.0f;

  for(int i = 0; i < ds/blockDim.x; i++) sdata[idx] += A[ds*blockIdx.x + i*blockDim.x + idx];
  
  for(int s = blockDim.x/2; s > 0; s/=2){
    __syncthreads();
    if (idx < s) sdata[idx] += sdata[idx + s];
  }
  
  if (idx == 0) sums[blockIdx.x] = sdata[0];

}


__global__ void softmax(float *A, float *sums, size_t ds){

  int idx = threadIdx.x;
  __shared__ float sdata[block_size];
  sdata[idx] = 0.0f;
  float val;

  for(int i = 0; i < ds/blockDim.x; i++){
    val = expf(A[ds*blockIdx.x + i*blockDim.x + idx]);
    A[ds*blockIdx.x + i*blockDim.x + idx] = val;
    sdata[idx] += val;
  }

  for(int s = blockDim.x/2; s > 0; s/=2){
    __syncthreads();
    if (idx < s) sdata[idx] += sdata[idx + s];
  }
  
  if (idx == 0) sums[blockIdx.x] = sdata[0];
  
  for(int i = 0; i < ds/blockDim.x; i++) A[ds*blockIdx.x + i*blockDim.x + idx] /= sdata[0];
  

}

bool validate(float *data, size_t sz){
  
  for (size_t i = 0; i < sz; i++){
    // printf("%f\n", expf(0.005)*(float)sz);
    float val = expf(element_val)*(float)sz;
    if (data[i] - val > 0.001) {printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], val); return false;}
  }
    return true;
}
int main(){

  float *h_A, *d_A, *d_sums;
  h_A = new float[DSIZE*DSIZE];  // allocate space for data in host memory
  // h_sums = new float[DSIZE]();
  for (int i = 0; i < DSIZE*DSIZE; i++)  // initialize matrix in host memory
    h_A[i] = element_val;
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sums, DSIZE*sizeof(float));  // allocate device space for vector d_sums

  cudaCheckErrors("hipMalloc failure"); // error checking
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  //row_sums<<<DSIZE, block_size>>>(d_A, d_sums, DSIZE);
  softmax<<<DSIZE, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");

  // cudaMemcpy(h_sums, d_sums, DSIZE*sizeof(float), cudaMemcpyDeviceToHost);
  // cudaCheckErrors("1 kernel execution failure or cudaMemcpy H2D failure");

  // if (!validate(h_sums, DSIZE)) return -1; 
  // printf("row sums correct!\n");

  hipMemcpy(h_A, d_A, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy D2H failure");

  printf("%.15f\n", h_A[312192]);

  for(int i = 0; i < DSIZE*DSIZE; i++){
    // printf("%f\n %f\n", h_A[i], 1/(float)DSIZE);
    if(h_A[i] - 1/(float)DSIZE > 0.00001
    ) {printf("results mismatch at %d, was: %.10f, should be: %.10f\n", i, h_A[i], 1/float(DSIZE)); return -1;}
  }
    printf("softmax correct!\n");
    


  return 0;
}
  
