
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


// const int N = 2048;      // matrix side dimension
const int Dim = 768;
const int block_size = 32;  // CUDA maximum is 1024 *total* threads in block  
const int block_size_linear = 768;
const int N_Layers = 12;
const int Vocab = 30522;
const int num_heads = 12;


__global__ void softmax_max(float *A, size_t ds) {

  int idx = threadIdx.x;
  __shared__ float sdata[block_size];
  sdata[idx] = 0.0f;
  float val = 0.0f;

  // Total elements this block is supposed to handle
  int total_elements = ds * ds;
  int start_index = blockIdx.x * ds; // Start index for this block
  int end_index = start_index + ds;  // End index for this block

  __shared__ float max_val;
  max_val = 0.0f;

  // Find the maximum value in the block

  for (int index = start_index + idx; index < end_index; index += blockDim.x) {
    if (index < ds*ds) sdata[idx] = max(A[index], sdata[idx]);
  }

  for(int s = blockDim.x/2; s > 0; s/=2){
    __syncthreads();
    if (idx < s) sdata[idx] = max(sdata[idx], sdata[idx + s]);
  }
  __syncthreads();

  if (idx == 0) max_val = sdata[0];
  __syncthreads();

  sdata[idx] = 0.0f;

  // Process elements
  for (int index = start_index + idx; index < end_index; index += blockDim.x) {
    if (index < total_elements) {
      val = expf(A[index] - max_val);
      A[index] = val;
      atomicAdd(&sdata[idx], val);
    }
  }

  __syncthreads();

  // Sum reduction in shared memory
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (idx < s) {
      sdata[idx] += sdata[idx + s];
    }
    __syncthreads();
  }

  // Normalize the values
  for (int index = start_index + idx; index < end_index; index += blockDim.x) {
    if (index < total_elements) {
      A[index] /= sdata[0];
    }
  }
}

__global__ void layernorm(float *A, float *B, int dim, float *gamma, float *beta){

    int idx = threadIdx.x;

    __shared__ float sdata[block_size];
    __shared__ float sum;
    sdata[idx] = 0.0f;
    sum = 0.0f;
    float val = 0.0f;

    for(int i = 0; i < dim/blockDim.x; i++) sdata[idx] += A[dim*blockIdx.x + i*blockDim.x + idx];

    for(int s = blockDim.x/2; s > 0; s/=2){
    __syncthreads();
    if (idx < s) sdata[idx] += sdata[idx + s];
    }

    if(idx == 0) sum = sdata[0]/dim;

    __syncthreads();

    sdata[idx] = 0.0f;

    for(int i = 0; i < dim/blockDim.x; i++){
        val = (A[dim*blockIdx.x + i*blockDim.x + idx] - sum);
        sdata[idx] += val*val;
    }

    for(int s = blockDim.x/2; s > 0; s/=2){
        __syncthreads();
        if (idx < s) sdata[idx] += sdata[idx + s];
    }

    if (idx == 0) sdata[0] = 1/sqrt(sdata[0]/dim + 0.00001);

    __syncthreads();

    for(int i = 0; i < dim/blockDim.x; i++){
        B[dim*blockIdx.x + i*blockDim.x + idx] = (A[dim*blockIdx.x + i*blockDim.x + idx] - sum)*sdata[0]*gamma[i*blockDim.x + idx] + beta[i*blockDim.x + idx];
    }

}

__global__ void matmul(const float *A, const float *B, float *C, int height, int width, int dim) {

  // declare cache in shared memory
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];

  int col = threadIdx.x+blockDim.x*blockIdx.x;
  int row = threadIdx.y+blockDim.y*blockIdx.y;

  if ((row < height) && (col < width)){
    float temp = 0;
    for (int i = 0; i < dim/block_size; i++) {

      // Load data into shared memory
      As[threadIdx.y][threadIdx.x] = A[row*dim + (block_size*i + threadIdx.x)];
      Bs[threadIdx.y][threadIdx.x] = B[col + width*(block_size*i + threadIdx.y)];

      __syncthreads();

      // Keep track of the running sum
      for (int k = 0; k < block_size; k++)
      	temp += As[threadIdx.y][k] * Bs[k][threadIdx.x]; // dot product of row and column
    
      __syncthreads();

    }

    C[row*width+col] = temp;
    if (row == 0 && col == 0) printf("%f\n", temp);
  }
}

__global__ void matmul_bias(const float *A, const float *B, float *C, float *bias, int height, int width, int dim, int N_tokens) {

  // declare cache in shared memory
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];

  int row = threadIdx.y+blockDim.y*blockIdx.y;
  int col = threadIdx.x+blockDim.x*blockIdx.x;
  

  if ((row < height) && (col < width)){
    float temp = 0;
    for (int i = 0; i < dim/block_size; i++) {

      // Load data into shared memory
      As[threadIdx.y][threadIdx.x] = A[row*dim + (block_size*i + threadIdx.x)];
      Bs[threadIdx.y][threadIdx.x] = B[col + width*(block_size*i + threadIdx.y)];

      __syncthreads();

      // Keep track of the running sum
      for (int k = 0; k < block_size; k++)
      	temp += As[threadIdx.y][k] * Bs[k][threadIdx.x]; // dot product of row and column
      __syncthreads();
    }
    
    if (row == 0 && col == 0) printf("%f\n", temp);
    if (row == 0 && col == 0) printf("%f\n", temp + bias[col]);

    if(row < N_tokens) C[row*width+col] = temp + bias[col];
    else C[row*width+col] = temp;
  }
}


__global__ void QK_V(const float *QK, const float *V, float *C, int Dim, int N) {

  // declare cache in shared memory
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];
  
  int col = threadIdx.x+blockDim.x*blockIdx.x; // create thread x index
  int row = threadIdx.y+blockDim.y*blockIdx.y; // create thread y index

  if ((row < N) && (col < Dim)){
    float temp = 0, val, sum = 0;

    for (int i = 0; i < N/block_size; i++) {

      // Load data into shared memory
      As[threadIdx.y][threadIdx.x] = expf(QK[row*N + (block_size*i + threadIdx.x)]);
      Bs[threadIdx.y][threadIdx.x] = V[col + Dim*(block_size*i + threadIdx.y)];

      __syncthreads();

      for (int k = 0; k < block_size; k++){
        val = As[threadIdx.y][k];
      	temp +=  val * Bs[k][threadIdx.x]; // dot product of row and column
        sum+=val;
      }

      __syncthreads();

    }

    // Write to global memory
    C[row*Dim+col] = temp/sum;
  }
}

__global__ void gelu(float *A, int dim){

    int idx = threadIdx.x;
    float x;

    for(int i = 0; i < dim/blockDim.x; i++){
        x = A[dim*blockIdx.x + i*blockDim.x + idx];
        A[dim*blockIdx.x + i*blockDim.x + idx] = x*0.5*(1.0 + tanhf(0.7978845608*(x + 0.044715*x*x*x)));
        // A[dim*blockIdx.x + i*blockDim.x + idx] = x/(1 + expf(-1.702*x));
    }
}

__global__ void add(float *A, float *B, int dim){

    int idx = threadIdx.x;

    for(int i = 0; i < dim/blockDim.x; i++){
        B[dim*blockIdx.x + i*blockDim.x + idx] = A[dim*blockIdx.x + i*blockDim.x + idx] + B[dim*blockIdx.x + i*blockDim.x + idx];
    }

}

// scale<<<N, block_size>>>(d_QK, head_dim, N, head_dim);

__global__ void scale(float *A, int N, int head_dim){

    int idx = threadIdx.x;

    for(int i = 0; i < N/blockDim.x; i++){
        A[N*blockIdx.x + i*blockDim.x + idx] = A[N*blockIdx.x + i*blockDim.x + idx]/sqrtf(head_dim);
    }

}

// set traingle values  and values outside N_tokens*N_tokens to -infinity
__global__ void set_inf(float *A, int dim, int N, int N_tokens){

    int idx = threadIdx.x;

    if(blockIdx.x < N_tokens){
      for(int i = 0; i < dim/blockDim.x; i++){
          if (i*blockDim.x + idx < N_tokens &&  i*blockDim.x + idx < blockIdx.x+1) continue;
          A[dim*blockIdx.x + i*blockDim.x + idx] = -100;
        }
    }
    else{
      for(int i = 0; i < dim/blockDim.x; i++) A[dim*blockIdx.x + i*blockDim.x + idx] = -100;
    }
}

// set all values to -infinity except for N_tokens*N_tokens block
__global__ void set_zero(float *A, int dim, int N, int N_tokens){

    int idx = threadIdx.x;

    if(blockIdx.x < N_tokens){
      for(int i = 0; i < dim/blockDim.x; i++){
          if (i*blockDim.x + idx < N_tokens) continue;
          A[dim*blockIdx.x + i*blockDim.x + idx] = 0;
        }
    }
    else{
      for(int i = 0; i < dim/blockDim.x; i++) A[dim*blockIdx.x + i*blockDim.x + idx] = 0;
    }
}

__global__ void isnan_test(float *data, int width, int height){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  while (idx < width){
    for (int i = 0; i < height; i++){
      if (isnan(data[(i*width) + idx])){
        printf("NAN at %d, %d\n", i, idx);
        return;
      }
    }
    idx += gridDim.x+blockDim.x;
    }
}

__global__ void matmul_mha_transpose(const float *A, const float *B, float *C, int height, int width, int dim, int head_dim, int head_num) {

  // declare cache in shared memory
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];

  int col = threadIdx.x+blockDim.x*blockIdx.x;
  int row = threadIdx.y+blockDim.y*blockIdx.y;

  if ((row < height) && (col < width)){
    float temp = 0;
    for (int i = 0; i < head_dim/block_size; i++) {

      // Load data into shared memory
      As[threadIdx.y][threadIdx.x] = A[row*dim + (block_size*i + threadIdx.x + head_dim*head_num)];
      Bs[threadIdx.y][threadIdx.x] = B[col*dim + (block_size*i + threadIdx.y + head_dim*head_num)];

      __syncthreads();

      // Keep track of the running sum
      for (int k = 0; k < block_size; k++)
      	temp += As[threadIdx.y][k] * Bs[k][threadIdx.x]; // dot product of row and column
    
      __syncthreads();

    }
    C[row*width+col] = temp;
  }
}

// // print val

// __global__ void print_val(float *data){
  
//     if(threadIdx.x == 0) printf("%f\n", data[0]);
  
// }

__global__ void matmul_mha(const float *A, const float *B, float *C, int height, int width, int dim, int head_dim, int head_num, int width_full ) {

// N, head_dim, N, head_dim, i, Dim

  // declare cache in shared memory
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];

  int row = threadIdx.y+blockDim.y*blockIdx.y;
  int col = threadIdx.x+blockDim.x*blockIdx.x + head_dim*head_num;
  // if(threadIdx.x == 0 && threadIdx.y == 0) printf("headnum, %d %d\n", row, head_dim*head_num);
  
  if ((row < height) && (col < width_full)){

    float temp = 0;
    for (int i = 0; i < dim/block_size; i++) {

      // Load data into shared memory
      As[threadIdx.y][threadIdx.x] = A[row*dim + (block_size*i + threadIdx.x)];
      Bs[threadIdx.y][threadIdx.x] = B[col + width_full*(block_size*i + threadIdx.y)];

      __syncthreads();

      // Keep track of the running sum
      for (int k = 0; k < block_size; k++)
      	temp += As[threadIdx.y][k] * Bs[k][threadIdx.x]; // dot product of row and column
    
      __syncthreads();

    }
    // if(
    //   threadIdx.x == 0 && threadIdx.y == 0 )printf("final %d %d %d\n", row*width_full, width_full,  col);
    C[row*width_full+col] = temp;
  }
}

int MHA(float *d_input, float *d_Q, float *d_K, float *d_V, float *d_QK, float *d_act, float *d_act_wide,
      float *linear[4], float *bias[4], float *ln[], float *mlp1, float *mlp_bias1, float *mlp2, float *mlp_bias2,
      int Dim, int N, int N_tokens, float *h_output, float *h_test, float *d_act2){

    dim3 threads(block_size, block_size);
    dim3 grid((Dim + threads.y - 1)/block_size, (N + threads.x - 1)/block_size);


    printf("Layer Normalization\n");
    layernorm<<<N_tokens, block_size>>>(d_input, d_act, Dim, ln[0], ln[1]);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_act, Dim, N);
    hipDeviceSynchronize();

    // calculate Q,K,V
    printf("Q\n");
    matmul_bias<<<grid, threads>>>(d_act, linear[0], d_Q, bias[0], N, Dim, Dim, N_tokens);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_Q, Dim, N);
    hipDeviceSynchronize();


    printf("K\n");
    matmul_bias<<<grid, threads>>>(d_act, linear[1], d_K, bias[1], N, Dim, Dim, N_tokens);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_K, Dim, N);
    hipDeviceSynchronize();

    // cudaMemcpy(h_output, d_K, N*Dim*sizeof(float), cudaMemcpyDeviceToHost);
    // cudaCheckErrors("cudaMemcpy D2H failure");

    // for(int k = 0; k < N_tokens; k++){
    //   for(int j = 0; j < Dim; j++)printf("%f ", h_output[k*Dim + j]);
    //   printf("\n");
    // }
    

    printf("V\n");
    matmul_bias<<<grid, threads>>>(d_act, linear[2], d_V, bias[2], N, Dim, Dim, N_tokens);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_V, Dim, N);
    hipDeviceSynchronize();

    int head_dim = Dim/num_heads;
    dim3 grid_mha((head_dim + threads.x - 1)/block_size, (N + threads.y - 1)/block_size);
    // printf("grid %d, %d\n", grid_mha.x, grid_mha.y);

    for (int i = 0; i < num_heads; i++){

      // printf("For loop %d\n", i);

      // Calculate QK.T
      // printf("QK\n"); 
      // printf("grid_mha %d, %d\n", grid_mha.x, grid_mha.y);
      matmul_mha_transpose<<<grid_mha, threads>>>(d_Q, d_K, d_QK, N, N, Dim, head_dim, i);
      cudaCheckErrors("kernel launch failure");
      isnan_test<<<1, 1>>>(d_QK, N, N);
      hipDeviceSynchronize();

      // cudaMemcpy(h_test, d_QK, N*N*sizeof(float), cudaMemcpyDeviceToHost);
      // cudaCheckErrors("cudaMemcpy D2H failure");

      // for(int k = 0; k < 6; k++){
      //   for(int j = 0; j < 6; j++)printf("%f ", h_test[k*32 + j]);
      //   printf("\n");
      // }

      // scale by sqrt(d_k)
      // printf("Scale\n");
      scale<<<N, block_size>>>(d_QK, N, head_dim);
      cudaCheckErrors("kernel launch failure");
      isnan_test<<<1, 1>>>(d_QK, N, N);
      hipDeviceSynchronize();

      // Set non tokens to -infinity
      // printf("Set non tokens to -infinity\n");
      set_inf<<<N, block_size>>>(d_QK, N, N, N_tokens);
      cudaCheckErrors("kernel launch failure");
      isnan_test<<<1, 1>>>(d_QK, N, N);
      hipDeviceSynchronize();

      // Softmax
      // printf("Softmax\n");
      softmax_max<<<N, block_size>>>(d_QK, N);
      cudaCheckErrors("kernel launch failure");
      isnan_test<<<1, 1>>>(d_QK, N, N);
      hipDeviceSynchronize();

      // Set non tokens to -infinity
      // printf("Set non tokens to -infinity\n");
      set_zero<<<N, block_size>>>(d_QK, N, N, N_tokens);
      cudaCheckErrors("kernel launch failure");
      isnan_test<<<1, 1>>>(d_QK, N, N);
      hipDeviceSynchronize();

      // printf("printing d_act\n");
      // print_val<<<1, 1>>>(d_act);
      // print_val<<<1, 1>>>(d_Q);
      // cudaDeviceSynchronize();
    
      // matmul
      // printf("QK_V\n");
      matmul_mha<<<grid_mha, threads>>>(d_QK, d_V, d_act, N, head_dim, N, head_dim, i, Dim);
      cudaCheckErrors("kernel launch failure");
      isnan_test<<<1, 1>>>(d_act, head_dim, N);
      hipDeviceSynchronize();

    }
    hipDeviceSynchronize();


    // cudaMemcpy(h_output, d_act, N*Dim*sizeof(float), cudaMemcpyDeviceToHost);
    // cudaCheckErrors("cudaMemcpy D2H failure");
    // for(int k = 0; k < 6; k++){
    //   for(int j = 0; j < 10; j++) printf("%.3f ", h_output[k*Dim + j]);
    //   printf("\n");
    // }


    // Calculate Final output
    printf("Final output\n");
    matmul_bias<<<grid, threads>>>(d_act, linear[3], d_act2, bias[3], N, Dim, Dim, N_tokens);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_act, Dim, N);
    hipDeviceSynchronize();
      

    // Residual connection
    // printf("Residual connection\n");
    add<<<N, block_size_linear>>>(d_act2, d_input, Dim);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_input, Dim, N);
    hipDeviceSynchronize();

    // Layer Normalization
    // printf("Layer Normalization\n");
    layernorm<<<N_tokens, block_size>>>(d_input, d_act, Dim, ln[2], ln[3]);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_input, Dim, N);
    hipDeviceSynchronize();

    // cudaMemcpy(h_test, d_input, N*Dim*sizeof(float), cudaMemcpyDeviceToHost);
    // cudaCheckErrors("cudaMemcpy D2H failure");
    // for(int k = 0; k < 6; k++){
    //   for(int j = 0; j < 5; j++) printf("%.3f ", h_test[k*Dim + j]);
    //   printf("\n");
    // }

    dim3 grid_wide((4*Dim + threads.x - 1)/block_size, (N + threads.y - 1)/block_size);
    // Matmul
    printf("Mlp1\n");
    matmul_bias<<<grid_wide, threads>>>(d_act, mlp1, d_act_wide, mlp_bias1, N, 4*Dim, Dim, N_tokens);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_act_wide, Dim, N);
    hipDeviceSynchronize();

    //gelu
    printf("Gelu\n");
    gelu<<<N, block_size>>>(d_act_wide, Dim);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_act, 4*Dim, N);
    hipDeviceSynchronize();

    // cudaMemcpy(h_output, d_act_wide, N*4*Dim*sizeof(float), cudaMemcpyDeviceToHost);
    // cudaCheckErrors("cudaMemcpy D2H failure");
    // for(int k = 0; k < 6; k++){
    //   for(int j = 0; j < 5; j++) printf("%.3f ", h_output[k*4*Dim + j]);
    //   printf("\n");
    // }

    // Matmul
    printf("mlp2\n");
    matmul_bias<<<grid, threads>>>(d_act_wide, mlp2, d_act, mlp_bias2, N, Dim, 4*Dim, N_tokens);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_act, Dim, N);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_act, N*Dim*sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");
    for(int k = 0; k < 6; k++){
      for(int j = 0; j < 5; j++) printf("%.3f ", h_output[k*Dim + j]);
      printf("\n");
    }


    // Residual connection
    printf("Residual connection\n");
    add<<<N, block_size_linear>>>(d_act, d_input, Dim);
    cudaCheckErrors("kernel launch failure");
    isnan_test<<<1, 1>>>(d_input, Dim, N);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_input, N*Dim*sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");
    for(int k = 0; k < 6; k++){
      for(int j = 0; j < 5; j++) printf("%.3f ", h_output[k*Dim + j]);
      printf("\n");
    }

    return 0;
    }
  
int Transformer(float *d_input, float *d_Q, float *d_K, float *d_V, float *d_QK, float *d_act, float *d_act_wide,
      float *linear[N_Layers][4], float *bias[N_Layers][4], float *ln[N_Layers][4], float *mlp1[N_Layers],
      float *mlp_bias1[N_Layers], float *mlp2[N_Layers], float *mlp_bias2[N_Layers], float *ln_final[2],
      float *proj_linear, float *d_output, int Dim, int N, int N_tokens, float *h_output, float *h_test, float *d_act2){


      for(int i = 0; i < 12; i++){
        printf("Layer %d\n", i);
        MHA(d_input, d_Q, d_K, d_V, d_QK, d_act, d_act_wide,
        linear[i], bias[i], ln[i], mlp1[i], mlp_bias1[i], mlp2[i], mlp_bias2[i],
         Dim, N, N_tokens, h_output, h_test, d_act2);
        hipDeviceSynchronize();
        printf("Layer %d done\n", i);
      }
      hipDeviceSynchronize();

      // Layer Normalization
      layernorm<<<N_tokens, block_size>>>(d_input, d_input, Dim, ln_final[0], ln_final[1]);
      cudaCheckErrors("kernel launch failure");

      dim3 threads(block_size, block_size);
      dim3 grid((Dim + block_size - 1)/block_size, (N + block_size - 1)/block_size);

      // Matmul
      matmul<<<grid, threads>>>(d_input, proj_linear, d_output, N, Vocab, Dim);
      cudaCheckErrors("kernel launch failure");

      printf("Done\n");
      return 0;

      }

int read_weight(float *arr, char *filename, int rows, int cols){

  printf("Reading %s\n", filename);
  
  FILE *file = fopen(filename, "rb");
  if (file == NULL) {
      printf("Error opening file\n");
      return 1;
  }

  fread(arr, sizeof(float), rows * cols, file);
  fclose(file);

  // for (int i = 0; i < rows; i++) {
  //     for (int j = 0; j < cols; j++) {
  //         // printf("%f ", arr[i * cols + j]);
  //         // check if the value is NaN
  //         if (isnan(arr[i * cols + j])) {
  //             printf("NaN detected at %d, %d\n", i, j);
  //             return 1;
  //         }
  //     }
  //     // printf("\n");
  // }

  return 0;
  
}

int main(){

    int N = 32;
    int N_tokens = 5;

    // declare host memory pointers

    float *h_input, *h_output, *h_linear[N_Layers][4], *h_bias[N_Layers][4], *h_ln[N_Layers][4],
           *h_mlp1[N_Layers], *h_mlp_bias1[N_Layers], *h_mlp2[N_Layers], *h_mlp_bias2[N_Layers],
           *h_final_ln[2], *h_proj_linear, *h_ans;

    float *d_input, *d_output, *d_Q, *d_K, *d_QK, *d_V, *d_ACT, *d_ACT_wide,
        *d_linear[N_Layers][4], *d_bias[N_Layers][4], *d_ln[N_Layers][4], *d_mlp1[N_Layers], 
        *d_mlp_bias1[N_Layers], *d_mlp2[N_Layers], *d_mlp_bias2[N_Layers], *d_final_ln[2], 
        *d_proj_linear, *d_act2;


    // allocate space for data in host memory

    h_input = new float[N*Dim];
    h_output = new float[N*Dim];
    h_ans = new float[N*Vocab];

    float* h_test = new float[Dim*N];

    for (int i = 0; i < N_Layers; i++){

      for (int j = 0; j < 4; j++){
        h_linear[i][j] = new float[Dim*Dim];
        h_bias[i][j] = new float[Dim];
        h_ln[i][j] = new float[Dim];
      }

      h_mlp1[i] = new float[Dim*4*Dim];
      h_mlp_bias1[i] = new float[4*Dim];
      h_mlp2[i] = new float[Dim*4*Dim];
      h_mlp_bias2[i] = new float[Dim];
    }

    for (int i = 0; i < 2; i++) h_final_ln[i] = new float[Dim];
    h_proj_linear = new float[Dim*Vocab];

    // init h_input to 0
    for (int i = 0; i < N*Dim; i++) h_input[i] = 0;

    // initialize matrix in host memory
    char filename[256];
    snprintf(filename, sizeof(filename), "gpt_weights/input.bin");
    read_weight(h_input, filename, N_tokens, Dim);


    for (int i = 0; i < N_Layers; i++){

      for(int j = 0; j < 2; j++){
        snprintf(filename, sizeof(filename), "gpt_weights/h.%d.ln_%d.weight.bin", i, j+1);
        read_weight(h_ln[i][j*2], filename, Dim, 1);
        snprintf(filename, sizeof(filename), "gpt_weights/h.%d.ln_%d.bias.bin", i, j+1);
        read_weight(h_ln[i][j*2+1], filename, Dim, 1);
      }
 
      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.attn.c_attn.weight.q.bin", i);
      read_weight(h_linear[i][0], filename, Dim, Dim);

      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.attn.c_attn.weight.k.bin", i);
      read_weight(h_linear[i][1], filename, Dim, Dim);

      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.attn.c_attn.weight.v.bin", i);
      read_weight(h_linear[i][2], filename, Dim, Dim);

      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.attn.c_proj.weight.bin", i);
      read_weight(h_linear[i][3], filename, Dim, Dim);

      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.attn.c_attn.bias.q.bin", i);
      read_weight(h_bias[i][0], filename, Dim, 1);
      
      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.attn.c_attn.bias.k.bin", i);
      read_weight(h_bias[i][1], filename, Dim, 1);

      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.attn.c_attn.bias.v.bin", i);
      read_weight(h_bias[i][2], filename, Dim, 1);

      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.attn.c_proj.bias.bin", i);
      read_weight(h_bias[i][3], filename, Dim, 1);


      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.mlp.c_fc.weight.bin", i);
      read_weight(h_mlp1[i], filename, Dim*4*Dim, 1);

      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.mlp.c_fc.bias.bin", i);
      read_weight(h_mlp_bias1[i], filename, 4*Dim, 1);

      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.mlp.c_proj.weight.bin", i);
      read_weight(h_mlp2[i], filename, Dim*4*Dim, 1);

      snprintf(filename, sizeof(filename), "gpt_weights/h.%d.mlp.c_proj.bias.bin", i);
      read_weight(h_mlp_bias2[i], filename, Dim, 1);
    }

    snprintf(filename, sizeof(filename), "gpt_weights/ln_f.weight.bin");
    read_weight(h_final_ln[0], filename, Dim, 1);

    snprintf(filename, sizeof(filename), "gpt_weights/ln_f.bias.bin");
    read_weight(h_final_ln[1], filename, Dim, 1);

    snprintf(filename, sizeof(filename), "gpt_weights/etw.weight.bin");
    read_weight(h_proj_linear, filename, Dim, Vocab);

    snprintf(filename, sizeof(filename), "gpt_weights/output.bin");
    read_weight(h_ans, filename, 5, Vocab);


    // allocate device space

    hipMalloc(&d_input, N*Dim*sizeof(float));
    hipMalloc(&d_output, N*Vocab*sizeof(float));
    hipMalloc(&d_Q, N*Dim*sizeof(float));
    hipMalloc(&d_K, N*Dim*sizeof(float));
    hipMalloc(&d_V, N*Dim*sizeof(float));  
    hipMalloc(&d_QK, N*N*sizeof(float));
    hipMalloc(&d_ACT, N*Dim*sizeof(float));
    hipMalloc(&d_ACT_wide, N*Vocab*sizeof(float));
    hipMalloc(&d_act2, N*Dim*sizeof(float));

    for (int i = 0; i < N_Layers; i++){

      for (int j = 0; j < 4; j++){
        hipMalloc(&d_linear[i][j], Dim*Dim*sizeof(float));
        hipMalloc(&d_bias[i][j], Dim*sizeof(float));
        hipMalloc(&d_ln[i][j], Dim*sizeof(float));
        cudaCheckErrors("hipMalloc failure"); // error checking
      }

      hipMalloc(&d_mlp1[i], Dim*4*Dim*sizeof(float));
      hipMalloc(&d_mlp_bias1[i], 4*Dim*sizeof(float));
      hipMalloc(&d_mlp2[i], Dim*4*Dim*sizeof(float));
      hipMalloc(&d_mlp_bias2[i], Dim*sizeof(float));
      cudaCheckErrors("hipMalloc failure"); // error checking
    }

    for (int i = 0; i < 2; i++) hipMalloc(&d_final_ln[i], Dim*sizeof(float));
    hipMalloc(&d_proj_linear, Dim*Vocab*sizeof(float));

    hipDeviceSynchronize();

    // copy data to device
    hipMemcpy(d_input, h_input, N*Dim*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    for (int i = 0; i < N_Layers; i++){

      for (int j = 0; j < 4; j++){
        hipMemcpy(d_bias[i][j], h_bias[i][j], Dim*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_linear[i][j], h_linear[i][j], Dim*Dim*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_ln[i][j], h_ln[i][j], Dim*sizeof(float), hipMemcpyHostToDevice);
        cudaCheckErrors("hipMemcpy H2D failure");
      }

      hipMemcpy(d_mlp1[i], h_mlp1[i], Dim*4*Dim*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_mlp_bias1[i], h_mlp_bias1[i], 4*Dim*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_mlp2[i], h_mlp2[i], Dim*4*Dim*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_mlp_bias2[i], h_mlp_bias2[i], Dim*sizeof(float), hipMemcpyHostToDevice);
      cudaCheckErrors("hipMemcpy H2D failure");
    }
    

    for (int i = 0; i < 2; i++){
      hipMemcpy(d_final_ln[i], h_final_ln[i], Dim*sizeof(float), hipMemcpyHostToDevice);
      cudaCheckErrors("hipMemcpy H2D failure");
    }

    hipMemcpy(d_proj_linear, h_proj_linear, Dim*Vocab*sizeof(float), hipMemcpyHostToDevice);

    // synchronize device
    hipDeviceSynchronize();

    // Launch kernel
    Transformer(d_input, d_Q, d_K, d_V, d_QK, d_ACT, d_ACT_wide, d_linear, d_bias,
     d_ln, d_mlp1, d_mlp_bias1, d_mlp2, d_mlp_bias2, d_final_ln, d_proj_linear, d_output, 
     Dim, N, N_tokens,h_output, h_test, d_act2);
    cudaCheckErrors("kernel launch failure");
    // N_tokens++;


    // synchronize device
    hipDeviceSynchronize();

    printf("Done\n");

    // // Copy results back to host
    // cudaMemcpy(h_output, d_output, 4*N*Dim*sizeof(float), cudaMemcpyDeviceToHost);
    // cudaCheckErrors("cudaMemcpy D2H failure");

    // // synchronize device
    // cudaDeviceSynchronize();
    // // Verify results
    // for (int i = 0; i < N*Vocab; i++){
    //   // printf("%d: %.10f: %.10f\n", i, h_output[i], h_ans[i]);
    //   if (h_output[i] - h_ans[i] > 0.001) {
    
    //     printf("results mismatch at %d, was: %.10f, should be: %.10f\n", i, h_output[i], h_ans[i]);
    //     return -1;
    //   }
    // }


    return 0;
}
  
